#include "hip/hip_runtime.h"
/*
Pseudocode:
char string;
char key -> string;
int value -> index; -> send to gpu to assign values 0-blah
thrust::stable_sort_by_key();
get sorted value
send to gpu -> sorted value, string
return string[value[tid]-1]
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include <iostream>
#include <string.h>

#define N 10
#define ARGC_EXPECTED_VAL 2

__global__ void fnKern(char *key, int *value, char *str)
{
	str[threadIdx.x] = key[value[threadIdx.x] - 1];
}

int main(int argc, char *argv[])
{
	if (argc != ARGC_EXPECTED_VAL)
	{
		std::cout << "Usage: sort_thrust <string to be sorted>\n";
		return 1;
	}
	thrust::device_vector<char> keyD(N, ' ');			//Size N, initialized with ' 's
	thrust::device_vector<char> strD(N, ' ');			//Size N, initialized with ' 's
	thrust::host_vector<char> keyH(N, ' ');				//The string to be sorted is taken from the command line
//	keyH = argv[1];
//	thrust::fill(keyH.begin(), keyH.end(), argv[1]);
	thrust::copy(argv[1], strlen(argv[1]), keyH.begin());
	thrust::copy(keyH.begin(), keyH.end(), keyD.begin());		//Copy the contents of keyH to keyD

	thrust::device_vector<int> valueD(N, 0);			//Size N, filled with 0s
	thrust::host_vector<int>valueH(N, 0);				//Host variable of size N
	thrust::sequence(valueD.begin(), valueD.end());			//set valueD's values to a sequence from 0 to N-1
//	thrust::host_vector<int> valueH(valueD.begin(), valueD.end());
//	thrust::stable_sort_by_key(keyD.begin(), keyD.end(), valueD);

	//ANIRUDH IS CONFUSED FROM HERE ON
	thrust::copy(valueD.begin(), valueD.end(), valueH.begin());	//Copy sorted values to host
	thrust::copy(valueH.begin(), valueH.end(), valueD.begin());	//Copy sorted values to device
	thrust::copy(keyH.begin(), keyH.end(), keyD.begin());		//Copy string to device
//	fnKern<<<1, N>>>(keyD, valueD, strD);
	return 0;
}
