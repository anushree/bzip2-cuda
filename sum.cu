
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
using namespace std; 
__global__ void fnKern(float *pA, float *pB, float *pSum) 
{ pSum[threadIdx.x] = pA[threadIdx.x] + pB[threadIdx.x]; } 
int main() 
{ int N = 512; 
int SIZE = N * sizeof(float); 
vector <float> a(N), b(N), sum(N); 
fill(a.begin(), a.end(), 100.123); 
fill(b.begin(), b.end(), 200.123); 
float *pA, *pB, *pSum; 
hipMalloc((void**)&pA, SIZE); 
hipMalloc((void**)&pB, SIZE); 
hipMalloc((void**)&pSum, SIZE); 
hipMemcpy(pA, &*a.begin(), SIZE, hipMemcpyHostToDevice); 
hipMemcpy(pB, &*b.begin(), SIZE, hipMemcpyHostToDevice); 
fnKern<<<1, N>>>(pA, pB, pSum); 
hipMemcpy(&*sum.begin(), pSum, SIZE, hipMemcpyDeviceToHost); 
for(int i = 1 ; i < N; ++i) 
{ if(sum[i] != sum[0]) 
{ cout << i << "->" << sum[i] << endl; 
return 0; } 
} 
cout << "Sum : " << sum[0] << endl; 
return 0; 
}
