#ifndef STRING_SORT_CU
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>

/*
#ifndef STRING_SORT_HEADER
#define STRING_SORT_HEADER
#endif
*/

//Taken from http://ldn.linuxfoundation.org/article/c-gpu-and-thrust-strings-gpu

class device_string
{
	public:
	int cstr_len;
	char* raw;
	thrust::device_ptr cstr;

	static char* pool_raw;
	static thrust::device_ptr pool_cstr;
	static thrust::device_ptr pool_top;

	__host__ static void init()
	{
		static bool v = true;
		if( v )
		{
			v = false;

			const int POOL_SZ = 10*1024*1024;

			pool_cstr = thrust::device_malloc(POOL_SZ);
			pool_raw  = raw_pointer_cast( pool_cstr );
			pool_top = pool_cstr;
		}
	}
	__host__ static void fini()
	{
		init();
		thrust::device_free(pool_cstr);
	}

	__host__ device_string( const device_string& s )
	{
		cstr_len = s.cstr_len;
		raw = s.raw;
		cstr = s.cstr;
	}

	__host__ device_string( const std::string& s ) : cstr_len( s.length() )
	{
		init();

		cstr = pool_top;
		pool_top += cstr_len+1;
		raw = raw_pointer_cast( cstr );

		hipMemcpy( raw, s.c_str(), cstr_len+1, hipMemcpyHostToDevice );
	}
	__host__ __device__ device_string() : cstr_len( -1 ), raw( 0 )
	{}

	__host__ operator std::string ()
	{
	std::string ret;
	thrust::copy( cstr, cstr+cstr_len, back_inserter(ret));
	return ret;
	}
};


char* device_string::pool_raw;
thrust::device_ptr device_string::pool_cstr;
thrust::device_ptr device_string::pool_top;

#endif
