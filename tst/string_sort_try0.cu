//#ifndef STRING_SORT_CU

//Taken from http://ldn.linuxfoundation.org/article/c-gpu-and-thrust-strings-gpu
//Also, https://groups.google.com/group/thrust-users/msg/0eac80d2e41cbcfb?pli=1, https://groups.google.com/group/thrust-users/browse_thread/thread/f4b1b825cc927df9?pli=1, 

#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include <cstring>
#include <vector>
#include <iterator>

//#include <hip/hip_runtime.h>

#define POOL_SZ (10*1024*1024)

using namespace std;

class device_string
{
public:
	int cstr_len;
	char* raw;
	thrust::device_ptr<char> cstr;

	static char* pool_raw;
	static thrust::device_ptr<char> pool_cstr;
	static thrust::device_ptr<char> pool_top;

	// Sets the variables up the first time its used.
	__host__ static void init()
	{
    		static bool v = true;
		if( v )
		{
			v = false;

			pool_cstr = thrust::device_malloc(POOL_SZ);
			pool_raw  = (char*)raw_pointer_cast( pool_cstr );
			pool_top = pool_cstr;
		}
        }
	// Destructor for device variables used.
	__host__ static void fin()
	{
		init();
		thrust::device_free(pool_cstr);
	}

	// Parametrized constructor to copy one device_string to another.
	__host__ device_string( const device_string& s )
	{
		cstr_len = s.cstr_len;
		raw = s.raw;
		cstr = s.cstr;
	}

	// Parametrized constructor to copy a std::string to device_string type
	__host__ device_string( const std::string& s )
	{
		cstr_len = s.length();
		init();
		cstr = pool_top;
		pool_top += cstr_len+1;
		raw = (char *) raw_pointer_cast(cstr);
		hipMemcpy( raw, s.c_str(), cstr_len+1, hipMemcpyHostToDevice );
	}

	// Default constructor.
	__host__ __device__ device_string()
	{
		cstr_len = -1;
		raw = NULL;
	}

	// Conversion operator to copy device_string type to std::string
	// This is where the problem is

	__host__ operator std::string()
	{
		std::string ret;
		//device_ptr<char*>::iterator it = cstr.begin();
		thrust::copy(cstr, cstr+cstr_len, back_inserter(ret));
		return ret;
	}
};

char* device_string::pool_raw;
thrust::device_ptr<char> device_string::pool_cstr;
thrust::device_ptr<char> device_string::pool_top;

// User-defined comparison operator
bool __device__ operator< (device_string lhs, device_string rhs)
{
	char *l = lhs.raw;
	char *r = rhs.raw;

	for( ; *l && *r && *l==*r; )
	{
	++l;
	++r;
	}
	return *l < *r;
}

int main()
{
	char* all_repeats_h = "abcb\0bcba\0cbab\0babc";
	int max_width = 4;

	vector<std::string> h_vec;

	for (int i = 0; i < max_width; i++)
	{
		h_vec.push_back(all_repeats_h + i*(max_width+1)*sizeof(char));
	}

	std::cout << "Content of h_vec..\n";
	for(int i = 0; i<h_vec.size(); i++)
	{
		std::cout << h_vec[i] << endl;
	}

	thrust::device_vector<device_string> d_vec;
	d_vec.reserve(h_vec.size());

	for(vector<std::string>::iterator iter = h_vec.begin(); iter!=h_vec.end(); ++iter)
	{
		device_string d_str(*iter);
		d_vec.push_back(d_str);
	}

	thrust::sort(d_vec.begin(), d_vec.end() );

	std::cout << " Done with sort().. \nThe sorted list of conjugates are: \n\n";
	for(int i = 0; i < d_vec.size(); i++)
	{
		std::string temp;
		temp = d_vec[i];
		std::cout << temp << endl;
	}
	return 0;
} 

//#endif
