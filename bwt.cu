#include <stdio.h>
#include <stdlib.h>

#include "bitonic_kernel.cu"

void output(int val[], int n)
{
	printf("%s", "Output:\n");
	
	for (int i = 0; i < n; i++)
	{
		printf("%d\t", val[i]);
	}
	printf("%s", "\n");
}

int main(int argc, char** argv)
{

    int values[NUM];

    for(int i = 0; i < NUM; i++)
    {
        values[i] = NUM - i;
    }
    
    output(values, NUM);

    int * dvalues;
    hipMalloc((void**)&dvalues, sizeof(int) * NUM);
    hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);

    bitonicSort<<<1, NUM, sizeof(int) * NUM>>>(dvalues);

	hipMemcpy(values, dvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost);

    hipFree(dvalues);
    
    output(values, NUM);

    bool passed = true;
    for(int i = 1; i < NUM; i++)
    {
        if (values[i-1] > values[i])
        {
            passed = false;
        }
    }

    printf( "Test %s\n", passed ? "PASSED" : "FAILED");

    return 0;
}
