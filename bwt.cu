
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//#include "bitonic_kernel.cu"

#define NUM    5

__device__ inline void swap(int & a, int & b)
{
	// Alternative swap doesn't use a temporary register:
	// a ^= b;
	// b ^= a;
	// a ^= b;
	
    int tmp = a;
    a = b;
    b = tmp;
}

__global__ static void bitonicSort(int * values)
{
    extern __shared__ int shared[];

    const int tid = threadIdx.x;

    // Copy input to shared mem.
    shared[tid] = values[tid];

    __syncthreads();

    // Parallel bitonic sort.
    for (int k = 2; k <= NUM; k *= 2)
    {
        // Bitonic merge:
        for (int j = k / 2; j>0; j /= 2)
        {
            int ixj = tid ^ j;
            
            if (ixj > tid)
            {
                if ((tid & k) == 0)
                {
                    if (shared[tid] > shared[ixj])
                    {
                        //swap(shared[tid], shared[ixj]);
                        int tmp = shared[tid];
    					shared[tid] = shared[ixj];
					    shared[ixj] = tmp;
                    }
                }
                else
                {
                    if (shared[tid] < shared[ixj])
                    {
                        //swap(shared[tid], shared[ixj]);
                        int tmp = shared[tid];
    					shared[tid] = shared[ixj];
					    shared[ixj] = tmp;
                    }
                }
            }
            
            __syncthreads();
        }
    }
    
    // Write result.
    values[tid] = shared[tid];
    
//    values[tid] = 0;
}


void output(int val[], int n)
{
	printf("%s", "Output:\n");
	
	for (int i = 0; i < n; i++)
	{
		printf("%d\t", val[i]);
	}
	printf("%s", "\n");
}

int main(int argc, char** argv)
{

    int values[NUM];

    for(int i = 0; i < NUM; i++)
    {
        values[i] = NUM - i;
    }
    
    output(values, NUM);

    int * dvalues;
    hipMalloc((void**)&dvalues, sizeof(int) * NUM);
    hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);

    bitonicSort<<<1, NUM, sizeof(int) * NUM>>>(dvalues);

	hipMemcpy(values, dvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost);

    hipFree(dvalues);
    
    output(values, NUM);

    bool passed = true;
    for(int i = 1; i < NUM; i++)
    {
        if (values[i-1] > values[i])
        {
            passed = false;
        }
    }

    printf( "Test %s\n", passed ? "PASSED" : "FAILED");

    return 0;
}
