#include "hip/hip_runtime.h"
#include <stdio.h>
# include "cutil.h"

__global__ void fnKern(int *pA, int *pB, int *pSum)
{
    pSum[threadIdx.x] = pA[threadIdx.x] + pB[threadIdx.x];
}

int main()
{
    const int N = 512;
    int SIZE = N * sizeof(float);
    int a[N], b[N], sum[N], aa[N], bb[N];

    for (int i = 0; i < N; i++)
    {
    	a[i] = b[i] = 100;
    	//sum[i] = 0;
    }
    
    printf("A: %d\nB: %d\n", a[0], b[0]);

    int *pA, *pB, *pSum;
    CUDA_SAFE_CALL(hipMalloc((void**)&pA, SIZE));
    CUDA_SAFE_CALL(hipMalloc((void**)&pB, SIZE));
    CUDA_SAFE_CALL(hipMalloc((void**)&pSum, SIZE));

    CUDA_SAFE_CALL(hipMemcpy(pA, a, SIZE, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(pB, b, SIZE, hipMemcpyHostToDevice));
    //hipMemcpy(pSum, sum, SIZE, hipMemcpyHostToDevice);

    fnKern<<<1, N>>>(pA, pB, pSum);
    CUT_CHECK_ERROR("fnKern failed");

    CUDA_SAFE_CALL(hipMemcpy(aa, pA, SIZE, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(bb, pB, SIZE, hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemcpy(sum, pSum, SIZE, hipMemcpyDeviceToHost));

    printf("A: %d\nB: %d\nSum: %d\n", aa[0], bb[0], sum[0]);
    return 0;
}
