#include <stdio.h>
#include <stdlib.h>

#include "../lib/bitonic_kernel.cu"

void output(int val[], int n)
{
	printf("%s", "Output:\n");

	for (int i = 0; i < n; i++)
	{
		printf("%d\t", val[i]);
	}
	printf("%s", "\n");
}

int main(int argc, char** argv)
{

	int values[NUM];

	for(int i = 0; i < NUM; i++)
	{
		values[i] = rand();
		//values[i] = NUM - i -1;
	}

	output(values, NUM);

	int * dvalues;
	hipMalloc((void**)&dvalues, sizeof(int) * NUM);
	hipMemcpy(dvalues, values, sizeof(int) * NUM, hipMemcpyHostToDevice);

	bitonicSort<<<1, NUM, sizeof(int) * NUM>>>(dvalues);
	
	////
	hipError_t ERR;
	ERR = hipGetLastError();
	printf("Status: %s\n", hipGetErrorString (ERR));
	////
	
	hipMemcpy(values, dvalues, sizeof(int) * NUM, hipMemcpyDeviceToHost);

	hipFree(dvalues);

	output(values, NUM);

	bool passed = true;
	for(int i = 1; i < NUM; i++)
	{
		if (values[i-1] > values[i])
		{
			passed = false;
		}
	}

	printf( "Test %s\n", passed ? "PASSED" : "FAILED");

	return 0;
}
