#include "hip/hip_runtime.h"
/*
Pseudocode:
char string;
char key -> string;
int value -> index; -> send to gpu to assign values 0-blah
thrust::stable_sort_by_key();
get sorted value
send to gpu -> sorted value, string
return string[value[tid]-1]
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include<hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstring>
#include <string.h>

#define ARGC_EXPECTED_VAL 2

__global__ void fnKern(char *key, int *val, char *str)
{
	str[threadIdx.x] = key[val[threadIdx.x] - 1];
}

int main(int argc, char *argv[])
{
	if (argc != ARGC_EXPECTED_VAL)
	{
		std::cout << "Usage: sort_thrust <string to be sorted>\n";
		return 1;
	}
	
	int N = strlen(argv[1]);
	
	char *key, *str;
	int *val;
	char *strH = new(char);	
	int *value = new(int);

	strH[0] = argv[1][0];
	value[0] = 0;
	for (int i = 1; i < N; i++)
	{
		strH[i] = argv[1][N - i -1];
		value[i] = N - i - 1;
	}
	//strH = strrev(argv[1]);	
	
	hipMalloc((void**)&key, sizeof(char) * N);
	hipMalloc((void**)&val, sizeof(int) * N);
	hipMalloc((void**)&str, sizeof(char) * N);
	
	thrust::device_ptr<char> keyD(key);	
	thrust::device_ptr<char> strD(str);
	thrust::device_ptr<int> valD(val);
	
	thrust::copy(strH, strH + N - 1, keyD);
	//thrust::copy(argv[1], argv[1],keyD);
	//thrust::copy(strH, strH + N - 2, keyD + 1);	//The string to be sorted is taken from the command line.
							//We originally had keyH.begin() here, but I did not see the need for a keyH at all

	//thrust::sequence(valD, valD + N);	//set valD's values to a sequence from 0 to N-1
	thrust::copy(value, value + N - 1, valD);
	
	thrust::stable_sort_by_key(keyD, keyD + N - 1, valD);
	
	//thrust::copy(valD.begin(), valD.end(), valH.begin());	//Copy sorted values to host
	//thrust::copy(valH.begin(), valH.end(), valD.begin());	//Copy sorted values to device
	//Whuuut are these two lines??? Table tennis? :P
	
	fnKern<<<1, N>>>(key, val, str);
	
	hipMemcpy(strH, str, N, hipMemcpyDeviceToHost);
//	strH = thrust::raw_pointer_cast(strD);	
	for (int i = 0; i < N; i++)
	std::cout << strH[i] << std::endl;

	hipFree(key);
	hipFree(str);
	hipFree(val);
	
	return 0;
}
