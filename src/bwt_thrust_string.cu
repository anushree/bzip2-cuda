#include "hip/hip_runtime.h"
/*
Pseudocode:
char string;
char key -> string;
int value -> index; -> send to gpu to assign values 0-blah
thrust::stable_sort_by_key();
get sorted value
send to gpu -> sorted value, string
return string[value[tid]-1]
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include<hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstring>

#define ARGC_EXPECTED_VAL 2

__global__ void fnKern(char *key, int *val, char *str)
{
	str[threadIdx.x] = key[val[threadIdx.x] - 1];
}

int main(int argc, char *argv[])
{
	/*if (argc != ARGC_EXPECTED_VAL)
	{
		std::cout << "Usage: sort_thrust <string to be sorted>\n";
		return 1;
	}*/
	char *word = new(char);
	std::cin >> word;
	int N = strlen(word);

	char *strH = new(char);	
	int *value = NULL;
	value = new int[N];
	
	strH[0] = word[0];
	value[0] = 0;
	for (int i = 1; i < N; i++)
	{
		strH[i] = word[N - i];
		value[i] = N - i;
	}

	char *key, *str;
	int *val;
	
	hipMalloc((void**)&key, sizeof(char) * N);
	hipMalloc((void**)&val, sizeof(int) * N);
	hipMalloc((void**)&str, sizeof(char) * N);

	thrust::device_ptr<char> keyD(key);	
	thrust::device_ptr<char> strD(str);
	thrust::device_ptr<int> valD(val);
	
	thrust::copy(strH, strH + N, keyD);

	thrust::copy(value, value + N, valD);
	
	thrust::stable_sort_by_key(keyD, keyD + N, valD);
	
//	fnKern<<<1, N>>>(key, val, str);
	
	hipMemcpy(strH, key, N, hipMemcpyDeviceToHost);
	hipMemcpy(value, val, N, hipMemcpyDeviceToHost);
//	strH = thrust::raw_pointer_cast(strD);	
	std::cout <<strH << std::endl;
	for (int i = 0; i < N; i++)
	std::cout << value[i] << std::endl;

	hipFree(key);
	hipFree(str);
	hipFree(val);
	
	return 0;
}
