#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include<hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstring>


// Refer http://ldn.linuxfoundation.org/article/c-gpu-and-thrust-strings-gpu
int main(int argc, char *argv[])
{
	char *word = new(char);
	std::cin >> word;
	int N = strlen(word);
	int i;

	char *str, *rot;
	
	hipMalloc((void**)&str, sizeof(char) * (N + 1));
	hipMalloc((void**)&rot, sizeof(char) * ((N + 1) * (N + 1)));
	
	thrust::device_ptr<char> strD(str);
//	thrust::device_ptr<device_string> rotD(rot);
	thrust::device_ptr<char> rotD(rot);
	
	thrust::copy(word, word + N, strD);

	for (i = 0; i < N; i++)				//Rotations
	{						//Check indices. 90% wrong. :P
		thrust::copy(strD + i, strD + N, rotD + (i * N));
		thrust::copy(strD, strD + i, rotD + (i * N) + (N - i));
	}
	
	//How to sort strings?
//	thrust::sort(rotD, rotD + N);

	for (i = 0; i < N; i++)
	{
		hipMemcpy(word, rot + (i * N), N, hipMemcpyDeviceToHost);
		std::cout << word <<std::endl;
	}
	
	hipFree(str);
	hipFree(rot);
	
	return 0;
}
