#include "hip/hip_runtime.h"
/*
Pseudocode:
char string;
char key -> string;
int value -> index; -> send to gpu to assign values 0-blah
thrust::stable_sort_by_key();
get sorted value
send to gpu -> sorted value, string
return string[value[tid]-1]
*/

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

#include<hip/hip_runtime.h>

#include <stdio.h>
#include <iostream>
#include <cstring>

#define ARGC_EXPECTED_VAL 2

__global__ void fnKern(int *key, int *val, int *str)
{
	str[threadIdx.x] = key[val[threadIdx.x] - 1];
}

int main(int argc, char *argv[])
{
	if (argc != ARGC_EXPECTED_VAL)
	{
		std::cout << "Usage: sort_thrust <string to be sorted>\n";
		return 1;
	}
	
	int N = strlen(argv[1]) - 1;
	
	int *key, *str;
	int *val;
	
	hipMalloc((void**)&key, sizeof(int) * N);
	hipMalloc((void**)&val, sizeof(int) * N);
	hipMalloc((void**)&str, sizeof(int) * N);
	
	thrust::device_ptr<int> keyD(key);	
	thrust::device_ptr<int> strD(str);
	thrust::device_ptr<int> valD(val);
	
	thrust::copy(argv[1], argv[1] + N, keyD);	//The string to be sorted is taken from the command line.
							//We originally had keyH.begin() here, but I did not see the need for a keyH at all

	thrust::sequence(valD, valD + N);	//set valD's values to a sequence from 0 to N-1
	
	thrust::stable_sort_by_key(keyD, keyD + N, valD);
	
	//thrust::copy(valD.begin(), valD.end(), valH.begin());	//Copy sorted values to host
	//thrust::copy(valH.begin(), valH.end(), valD.begin());	//Copy sorted values to device
	//Whuuut are these two lines??? Table tennis? :P
	
	fnKern<<<1, N>>>(key, val, str);
	
	char *strH = new(char);		
	hipMemcpy(strH, str, N, hipMemcpyDeviceToHost);
//	strH = thrust::raw_pointer_cast(strD);	
	for (int i =0; i < N; i++)
	std::cout << strH[i] << std::endl;

	hipFree(key);
	hipFree(str);
	hipFree(val);
	
	return 0;
}
